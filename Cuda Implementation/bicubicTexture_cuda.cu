#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>



#include <hip/hip_vector_types.h>

// includes, cuda
#include <hip/hip_runtime_api.h>

typedef unsigned int uint;
typedef unsigned char uchar;

//#include "bicubicTexture_kernel.cuh"
#include "Ray.h"
#include "hitable.h"
#include "vec3.h"
#include "hitable_list.h"
#include "sphere.h"


extern "C" {
#include "Ray.h"
}

hipArray *d_imageArray = 0;

extern "C" void initTexture(int imageWidth, int imageHeight, uchar *h_data)
{
	hipResourceDesc texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));
	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_imageArray;
}

extern "C" void freeTexture()
{
	checkCudaErrors(hipFreeArray(d_imageArray));
}

__device__ static int ticks = 1;

__device__ vec3 castRay(const ray& r, hitable **world)
{	
	hit_record rec;
	if ((*world)->hit(r, 0.0, FLT_MAX, rec)) 
	{		 
		return 0.5f*vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
	}
	else 
	{
		vec3 unit_direction = unit_vector(r.direction());
		float t = 0.5f*(unit_direction.y() + 1.0f);
		return (1.0f - t)*vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
	}	
}

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>
__device__ const int particlecountDevice = 50 + 5;
const int particlecountHost = 50 + 5;
__device__ const float boundary = 50; // +/- x and y boundaries
hitable **d_list;
hitable **d_world;
int posXrand[particlecountHost] = {};
int posYrand[particlecountHost] = {};
int posZrand[particlecountHost] = {};
int velXrand[particlecountHost] = {};
int velYrand[particlecountHost] = {};
int velZrand[particlecountHost] = {};

__global__ void create_world(hitable **d_list, hitable **d_world, int *posX, int *posY, int *posZ, int *velX, int *velY, int *velZ) 
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		// Create borders (Walls etc. are just oversized spheres)
		d_list[0] = new sphere(vec3(0, 0, -10050), 10000, 0, 0, 0); // Rear wall
		d_list[1] = new sphere(vec3(-10050, 0, -3), 10000, 0, 0, 0); // Left wall
		d_list[2] = new sphere(vec3(10050, 0, -3), 10000, 0, 0, 0); // Right wall
		d_list[3] = new sphere(vec3(0, -10050, -1), 10000, 0, 0, 0); // Floor
		d_list[4] = new sphere(vec3(0, 10050, -1), 10000, 0, 0, 0); // Ceiling
		
		// Initialise spheres into empty space		
		for (int i = 5; i < particlecountDevice; i++) {
			if (velX[i] == 0 && velY[i] == 0 && velZ[i] == 0) {
				velX[i] += 1;
			}
			d_list[i] = new sphere(vec3(posX[i], posY[i], posZ[i]), 0.5, velX[i], velY[i], velZ[i]);
		}

		*d_world = new hitable_list(d_list, particlecountDevice);
	}
}

__global__ void free_world(hitable **d_list, hitable **d_world)
{
	for (int i = 0; i < particlecountDevice; i++) {
		delete d_list[i];
	}
	delete *d_world;
}

__global__ void updatePositions(hitable **d_list)
{
	uint particleindlist = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Indexes 0-4 are the Walls, floor and ceiling.
	if (particleindlist < 5)
		return;
	
	sphere* kernelSphere = (sphere*)(d_list[particleindlist]);
	float x = kernelSphere->center.x();
	float y = kernelSphere->center.y();
	float z = kernelSphere->center.z();

	// Retrieve object velocities
	float vx = kernelSphere->_vx;
	float vy = kernelSphere->_vy;
	float vz = kernelSphere->_vz;

	float newX = x + vx;
	float newY = y + vy;
	float newZ = z + vz;

	if (newX > boundary) {
		newX = boundary - (newX - boundary);
		vx = -vx; // Reverse velocity on offending axis due to rebound
	}
	else if (newX < -boundary) {
		newX = newX - (newX - -boundary);
		vx = -vx;
	}
	else if (newX == boundary || newX == -boundary)
		vx = -vx;

	if (newY > boundary) {
		newY = boundary - (newY - boundary);
		vy = -vy; // Reverse velocity on offending axis due to rebound
	}
	else if (newY < -boundary) {
		newY = newY - (newY - -boundary);
		vy = -vy;
	}
	else if (newY == boundary || newY == -boundary)
		vy = -vy;

	if (newZ < -boundary) {
		newZ = -boundary - (newZ - -boundary);
		vz = -vz; // Reverse velocity on offending axis due to rebound
	}
	else if (newZ > 0) {
		newZ = -newZ;
		vz = -vz;
	}
	else if (newZ == boundary || newZ == 0)
		vz = -vz;

	// Update position and velocities
	kernelSphere->setPos(newX, newY, newZ);
	kernelSphere->setVel(vx, vy, vz);

	// Re-assign object at pointer location
	d_list[particleindlist] = kernelSphere;
}

__global__ void gravityFall(hitable **d_list)
{
	uint particleindlist = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleindlist < 5)
		return;

	sphere* kernelSphere = (sphere*)(d_list[particleindlist]);
	float x = kernelSphere->center.x();
	float y = kernelSphere->center.y();
	float z = kernelSphere->center.z();

	float newY = y - 1;
	
	if (newY < -boundary)
		newY = boundary;

	kernelSphere->setPos(x, newY, z);
	d_list[particleindlist] = kernelSphere;
}

__global__ void d_render(uchar4 *d_output, uint width, uint height, hitable **d_world)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint i = y * width + x;

	// Labs/Lectures code
	float u = x / (float)width; //----> [0, 1]x[0, 1]
	float v = y / (float)height;
	u = 2.0*u - 1.0; //---> [-1, 1]x[-1, 1]
	v = -(2.0*v - 1.0);
	u *= width / (float)height;
	u *= 2.0;
	v *= 2.0;
	vec3 eye = vec3(0, 0.5, 1.5);
	float distFrEye2Img = 1.0;;
	if ((x < width) && (y < height))
	{
		//for each pixel
		vec3 pixelPos = vec3(u, v, eye.z() - distFrEye2Img);
		//fire a ray:
		ray r;
		r.rayOri = eye;
		r.rayDir = pixelPos - eye; //view direction along negtive z-axis!
		vec3 col = castRay(r, d_world);

		// Colours
		float red = col.x();
		float green = col.y();
		float blue = col.z();

		d_output[i] = make_uchar4(red * 255, green * 255, blue * 255, 0);
	}
}

#include <time.h>
extern "C" void initialWorld() {
	// Generate random numbers
	// Random positions arrays
	srand(time(NULL));
	for (int i = 0; i < particlecountHost; ++i) {
		posXrand[i] = rand() % 100;
		posXrand[i] -= 50;
	}
	srand(time(NULL));
	for (int i = 0; i < particlecountHost; ++i) {
		posYrand[i] = rand() % 200 - 100;
		posYrand[i] -= 50;
		while (posYrand[i] < -50) {
			posYrand[i] += 50;
		}
	}
	srand(time(NULL));
	for (int i = 0; i < particlecountHost; ++i) {
		posZrand[i] = rand() % 50;
		posZrand[i] -= 50;
	}

	// Random velocities arrays
	srand(time(NULL));
	for (int i = 0; i < particlecountHost; ++i) {
		velXrand[i] = rand() % 6;
		velXrand[i] -= 3;
	}
	srand(time(NULL));
	for (int i = 0; i < particlecountHost; ++i) {
		velYrand[i] = rand() % 8 - 5;
		while (velYrand[i] < -3) {
			velYrand[i] += 3;
		}
	}
	srand(time(NULL));
	for (int i = 0; i < particlecountHost; ++i) {
		velZrand[i] = rand() % 7 - 4;
		while (velZrand[i] < -3) {
			velZrand[i] += 3;
		}
	}

	// Allocate and fill GPU memory
	int *xrand = 0;
	int *yrand = 0;
	int *zrand = 0;
	int *vxrand = 0;
	int *vyrand = 0;
	int *vzrand = 0;
	checkCudaErrors(hipMalloc((void **)&xrand, particlecountHost * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&yrand, particlecountHost * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&zrand, particlecountHost * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&vxrand, particlecountHost * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&vyrand, particlecountHost * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&vzrand, particlecountHost * sizeof(int)));
	hipMemcpy(xrand, posXrand, particlecountHost * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(yrand, posYrand, particlecountHost * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(zrand, posZrand, particlecountHost * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(vxrand, velXrand, particlecountHost * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(vyrand, velYrand, particlecountHost * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(vzrand, velZrand, particlecountHost * sizeof(int), hipMemcpyHostToDevice);		
	
	// Allocate GPU memory
	checkCudaErrors(hipMalloc((void **)&d_list, particlecountHost * sizeof(hitable *)));
	checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *))); 

	create_world << <1, 1 >> > (d_list, d_world, xrand, yrand, zrand, vxrand, vyrand, vzrand);

	hipFree(xrand);
	hipFree(yrand);
	hipFree(zrand);
	hipFree(vxrand);
	hipFree(vyrand);
	hipFree(vzrand);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

// render image using CUDA
extern "C" void render(int width, int height, dim3 blockSize, dim3 gridSize, uchar4 *output)
{
	d_render <<<gridSize, blockSize>>> (output, width, height, d_world);
	getLastCudaError("kernel failed");
	//free_world << <1, 1 >> > (d_list, d_world);
}

extern "C" void positionsUpdate()
{
	updatePositions << <1, particlecountHost >> > (d_list); // 1 thread per particle
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

extern "C" void gravityEngaged() 
{
	gravityFall << <1, particlecountHost >> > (d_list); // 1 thread per particle
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) 
{
	if (result) 
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}


#endif
